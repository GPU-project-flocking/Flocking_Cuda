#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <hip/device_functions.h>

#include <crt/host_defines.h>

#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <string>
#include <chrono>


//global variables
float2* pos_dev;
float2* vel_dev;
float2* acc_dev;

float2* sep_dev;
float2* align_dev;
float2* cohesion_dev;

float2* pos_host;
float2* vel_host;

float2  averagePos;
float2 averageForward;

#define BlockSize 256
#define NBOIDS 5
#define FLOCKING_RAD 50.0f;
#define COHESION_STRENGTH 1.0f;
#define ALIGNMENT_STRENGTH 1.0f;
#define SEPARATION_STRENGTH 1.0f;
#define SAFE_RADIUS 5.0f;
#define MAX_SPEED 5.0f;

//vector math -- may update functions

__device__
float2 negateVector(float2 a) {
	float2 negated;
	negated.x = -a.x;
	negated.y = -a.y;
	return negated;
}

__device__
bool vector2dEquals(float2 a, float2 b) {
	if (a.x == b.x && a.y == b.y) {
		return true;
	} else {
		return false;
	}
}

__device__
float distance(float2 myPos, float2 theirPos) {
	float dx = myPos.x - theirPos.x;
	float dy = myPos.y - theirPos.y;

	float dist = sqrt(dx*dx + dy*dy);
	return dist;
}


__device__
float2 add2dVectors(float2 v1, float2 v2) {
	float2 temp = make_float2(v1.x, v1.y);
	temp.x += v2.x;
	temp.y += v2.y;
	return temp;
}

__device__
float2 sub2dVectors(float2 v1, float2 v2) {
	float2 temp = make_float2(v1.x, v1.y);
	temp.x -= v2.x;
	temp.y -= v2.y;
	return temp;
}

__device__
float2 mulVectorByScalar(float scalar, float2 vector) {
	// Temp to not overwrite original vector
	float2 temp = make_float2(vector.x, vector.y);
	temp.x *= scalar;
	temp.y *= scalar;
	return temp;
}

__device__
float2 divVectorByScalar(float scalar, float2 vector) {
	float2 temp = make_float2(vector.x, vector.y);
	temp.x /= scalar;
	temp.y /= scalar;
	return temp;
}

__device__
float calcLength(float2 vec) {
	return sqrt(vec.x * vec.x + vec.y * vec.y
	);
}

__device__
float2 normalizeVector(float2 vector) {
	float2 temp = make_float2(vector.x, vector.y);
	float length = calcLength(temp);
	if (length > 0) {
		temp.x /= length;
		temp.y /= length;
	}
	return temp;
}

//-----------------end vec funcs------------------------

__host__ void calc_average_forward() {
	int counter = 0;
	float2 sum = make_float2(0.0, 0.0);
	for (int i = 0; i < NBOIDS; i++) {
		sum.x += vel_host[i].x;
		sum.y += vel_host[i].y;
		counter++;
	}
	averageForward.x = sum.x / counter;
	averageForward.y = sum.y / counter;
}

__host__ void calc_average_pos() {
	int counter = 0;
	float2 sum = make_float2(0.0, 0.0);
	for (int i = 0; i < NBOIDS; i++) {
		sum.x += pos_host[i].x;
		sum.y += pos_host[i].y;
		counter++;
	}
	averagePos.x = sum.x / counter;
	averagePos.y = sum.y / counter;
}

__global__ void updatePos(int numBoids, float2* vel_dev, float2* pos_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		if (pos_dev[i].x > 10000.0f || pos_dev[i].y > 10000.0f) {
			pos_dev[i].x = 0;
			pos_dev[i].y = 0;
		}
		

		pos_dev[i] = add2dVectors(pos_dev[i], vel_dev[i]);
		//pos_dev[i] = newPos;
	}
}

__device__ float2 calc_separation_accel(int numBoids, float2* pos_dev, float2* vel_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	float safeDist = SAFE_RADIUS;
	safeDist = safeDist + safeDist;
	float separationStrength = SEPARATION_STRENGTH;
	float2 totalVel = make_float2(0.0f, 0.0f);

	if (i < numBoids) { 
		float2 boidPos = make_float2(pos_dev[i].x, pos_dev[i].y);
		float2 boidVel = make_float2(vel_dev[i].x, vel_dev[i].y);

		for (int i = 0; i < numBoids; i++) {
			//printf("%s", vector2dEquals(boidPos, pos_dev[i]) && vector2dEquals(boidVel, vel_dev[i]) ? "true\n" : "");
			float2 siblingPos = pos_dev[i];
			float2 siblingVel = vel_dev[i];
			//check to see if current boid is self
			if (vector2dEquals(boidPos, siblingPos) && vector2dEquals(boidVel, siblingVel)) {
				continue;
			}

			float2 accel = sub2dVectors(boidPos, siblingPos);
			float dist = calcLength(accel);
			//printf("%d ", pos_dev[i].x);
			//toggle int
			if (dist > 8.0f) {
				continue;
			}
			if (dist < safeDist) {
				accel = normalizeVector(accel);
				accel = divVectorByScalar(safeDist, mulVectorByScalar((safeDist - dist), accel));
				totalVel = add2dVectors(totalVel, accel);
			}
		}

		if (calcLength(totalVel) > 1) {
			totalVel = normalizeVector(totalVel);
		}

		return mulVectorByScalar(separationStrength, totalVel);
	}

	return make_float2(0.0f, 0.0f);
}

__device__ float2 calc_alignment_accel(int numBoids, float2 averageForward) {
	float maxSpeed = MAX_SPEED;
	float alignStr = ALIGNMENT_STRENGTH;

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		float2 accel = divVectorByScalar(maxSpeed, averageForward);
		if (calcLength(accel) > 1) {
			accel = normalizeVector(accel);
		}
		return mulVectorByScalar(alignStr, accel);
	}
	return make_float2(0.0f, 0.0f);
}

__device__ float2 calc_cohesion_accel(int numBoids, float2 averagePos, float2* pos_dev) {
	float flockRad = FLOCKING_RAD;
	float cohesionStr = COHESION_STRENGTH;

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		float2 accel = sub2dVectors(averagePos, pos_dev[i]);
		float dist = calcLength(pos_dev[i]);

		accel = normalizeVector(accel);

		if(dist < flockRad) {
			accel = mulVectorByScalar(dist, accel);
			accel = divVectorByScalar(flockRad, accel);
		}

		return mulVectorByScalar(cohesionStr, accel);
	}
	return make_float2(0.0f, 0.0f);
}

__global__
void generateInitialPosition(int numBoids, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		pos_dev[i].x = 0.0f;
		pos_dev[i].y = 0.0f;
		vel_dev[i].x = 0.0f;
		vel_dev[i].y = 0.0f;
		acc_dev[i].x = 0.0f;
		acc_dev[i].y = 0.0f;
		sep_dev[i].x = 0.0f;
		sep_dev[i].y = 0.0f;
		align_dev[i].x = 0.0f;
		align_dev[i].y = 0.0f;
		cohesion_dev[i].x = 0.0f;
		cohesion_dev[i].y = 0.0f;
		
	}
	
}

__host__
void startCuda(int numBoids) {
	//printf("\nDefining cuda variables\n");
	dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));

	// Malloc for device
	hipMalloc((void**)&pos_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&vel_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&acc_dev, numBoids * sizeof(float2));

	hipMalloc((void**)&sep_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&align_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&cohesion_dev, numBoids * sizeof(float2));

	//malloc for host
	pos_host = (float2*)malloc(numBoids * sizeof(float2));
	vel_host = (float2*)malloc(numBoids * sizeof(float2));

	//set random velocity
	for (int i = 0; i < numBoids; i++) {
		vel_host[i].x = ((float) rand() / (RAND_MAX));
		vel_host[i].y = ((float) rand() / (RAND_MAX));
	}
	
	// Setup Kernels
	//printf("\nGenerating initial position\n");
	generateInitialPosition<<<fullBlocksPerGrid, BlockSize>>>(numBoids, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);

	hipMemcpy(vel_dev, vel_host, numBoids * sizeof(float2), hipMemcpyHostToDevice);
	hipMemcpy(pos_host, pos_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);
	hipMemcpy(vel_host, vel_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);


	//for debugging
	/*printf("after\n");
	for (int i = 0; i < numBoids; i++) {
		printf("x = %f, y = %f\n", vel_host[i].x, vel_host[i].y);
	}*/
}

__global__
void update(int numBoids, float2 averagePos, float2 averageForward, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		//cohesion
		float2 cohesion = calc_cohesion_accel(numBoids, averagePos, pos_dev);
		//separation
		float2 separation = calc_separation_accel(numBoids, pos_dev, vel_dev);
		//alignment
		float2 alignment = calc_alignment_accel(numBoids, averageForward);
		
		//printf("cohesion: %f\nseparation: %f\nalignment: %f\n", cohesion, separation, alignment);

		vel_dev[i] = add2dVectors(vel_dev[i], cohesion);
		vel_dev[i] = add2dVectors(vel_dev[i], separation);
		vel_dev[i] = add2dVectors(vel_dev[i], alignment);

		if (calcLength(vel_dev[i]) > 50.0f) {
			vel_dev[i] = normalizeVector(vel_dev[i]);
			vel_dev[i] = mulVectorByScalar(50.0f, vel_dev[i]);
			//printf("%d ", calcLength(vel_dev[i]));	
		}
		if (calcLength(vel_dev[i]) < 0.0f) {
			vel_dev[i] = normalizeVector(vel_dev[i]);
			vel_dev[i] = mulVectorByScalar(50.0f, vel_dev[i]);
			//printf("%d ", calcLength(vel_dev[i]));	
		}
		//printf("%d ", calcLength(vel_dev[i]));	
	}
		
}

__host__
int main(int argc, char* argv[]) 
{
	using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

    auto t1 = high_resolution_clock::now();

	int numB = std::stoi(argv[1]);
	int iterations = std::stoi(argv[2]);

	

	dim3 fullBlocksPerGrid((int)ceil(float(numB) / float(BlockSize)));
	
	//int iterations = 1000;
	

  	startCuda(numB);

	//printf("\nRunning Simulation with %d boids and %d iterations\n", numB, iterations);
	for (int i = 0; i < iterations; i++) {
		
		hipMemcpy(vel_host, vel_dev, numB * sizeof(float2), hipMemcpyDeviceToHost);
		hipMemcpy(pos_host, pos_dev, numB * sizeof(float2), hipMemcpyDeviceToHost);
		calc_average_pos();
		calc_average_forward();
		
		
		update<<<fullBlocksPerGrid, BlockSize>>>(numB, averagePos, averageForward, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);
		updatePos<<<fullBlocksPerGrid, BlockSize>>>(numB, vel_dev, pos_dev);
		//for debugging will remove
		//cudaMemcpy(vel_host, vel_dev, numB * sizeof(float2), cudaMemcpyDeviceToHost);
		//cudaMemcpy(pos_host, pos_dev, numB * sizeof(float2), cudaMemcpyDeviceToHost);
		//printf("guy1-x: %f, guy1-y: %f | ", pos_host[0].x, pos_host[0].y);
		//printf("guy2-x: %f, guy2-y: %f\n", pos_host[1].x, pos_host[1].y);
	}

    
   hipFree(pos_dev);
   hipFree(vel_dev);
   hipFree(acc_dev);
   hipFree(sep_dev);
   hipFree(align_dev);
   hipFree(cohesion_dev);
	
   free(pos_host);

   auto t2 = high_resolution_clock::now();
   duration<double, std::milli> ms_double = t2 - t1;
   printf("%f", ms_double);

   return 0;
}
