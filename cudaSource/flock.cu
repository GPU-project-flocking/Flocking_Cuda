
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

//global variables
float2* pos_dev;
float2* vel_dev;
float2* acc_dev;

float2* sep_dev;
float2* align_dev;
float2* cohesion_dev;

float2* pos_host;
float2* vel_host;

float2  averagePos;
float2 averageForward;

#define BlockSize 256
#define NBOIDS 10
#define FLOCKING_RAD 50.0f;
#define COHESION_STRENGTH 5.0f;

/*****************************************************************
*
*	Vector Functions -- will update
*
****************************************************************/

__device__
float distanceFormula(float2 myPos, float2 theirPos) {
	float dx = myPos.x - theirPos.x;
	float dy = myPos.y - theirPos.y;

	float dist = sqrt(dx*dx + dy*dy);
	return dist;
}


__device__
float2 add2Vectors(float2 v1, float2 v2) {
	float2 temp = make_float2(v1.x, v1.y);
	temp.x += v2.x;
	temp.y += v2.y;
	return temp;
}

__device__
float2 sub2Vectors(float2 v1, float2 v2) {
	float2 temp = make_float2(v1.x, v1.y);
	temp.x -= v2.x;
	temp.y -= v2.y;
	return temp;
}

__device__
float2 mulVectorByScalar(float scalar, float2 vector) {
	// Temp to not overwrite original vector
	float2 temp = make_float2(vector.x, vector.y);
	temp.x *= scalar;
	temp.y *= scalar;
	return temp;
}

__device__
float2 divVectorByScalar(float scalar, float2 vector) {
	float2 temp = make_float2(vector.x, vector.y);
	temp.x /= scalar;
	temp.y /= scalar;
	return temp;
}

__device__
float magnitudeOfVector(float2 vector) {
	return sqrt(
		vector.x * vector.x +
		vector.y * vector.y
	);
}

__device__
float2 normalizeVector(float2 vector) {
	float2 temp = make_float2(vector.x, vector.y);
	float magnitude = magnitudeOfVector(temp);
	if (magnitude > 0) {
		temp.x /= magnitude;
		temp.y /= magnitude;
	}
	return temp;
}

//-----------------end vec funcs------------------------

__host__ void calc_average_forward() {
	int counter = 0;
	float2 sum = make_float2(0.0, 0.0);
	for (int i = 0; i < NBOIDS; i++) {
		sum.x += vel_host[i].x;
		sum.y += vel_host[i].y;
		counter++;
	}
	averageForward.x = sum.x / counter;
	averageForward.y = sum.y / counter;
}

__host__ void calc_average_pos() {
	int counter = 0;
	float2 sum = make_float2(0.0, 0.0);
	for (int i = 0; i < NBOIDS; i++) {
		sum.x += pos_host[i].x;
		sum.y += pos_host[i].y;
		counter++;
	}
	averagePos.x = sum.x / counter;
	averagePos.y = sum.y / counter;
}

__device__ void updatePos(int numBoids, float2* vel_dev, float2* pos_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		float2 newPos = add2Vectors(pos_dev[i], vel_dev[i]);
		pos_dev[i] = newPos;
	}
}

__device__ float2 calc_cohesion_accel(int numBoids, float2 averagePos, float2* pos_dev) {
	float flockRad = FLOCKING_RAD;
	float cohesionStr = COHESION_STRENGTH;

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		float2 accel = sub2Vectors(averagePos, pos_dev[i]);
		float dist = magnitudeOfVector(pos_dev[i]);

		accel = normalizeVector(accel);

		if(dist < flockRad) {
			accel = mulVectorByScalar(dist, accel);
			accel = divVectorByScalar(flockRad, accel);
		}

		return mulVectorByScalar(cohesionStr, accel);
	}
	return make_float2(0.0f, 0.0f);
}

__global__
void generateInitialPosition(int numBoids, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		pos_dev[i].x = 0.0f;
		pos_dev[i].y = 0.0f;
		vel_dev[i].x = 0.0f;
		vel_dev[i].y = 0.0f;
		acc_dev[i].x = 0.0f;
		acc_dev[i].y = 0.0f;
		sep_dev[i].x = 0.0f;
		sep_dev[i].y = 0.0f;
		align_dev[i].x = 0.0f;
		align_dev[i].y = 0.0f;
		cohesion_dev[i].x = 0.0f;
		cohesion_dev[i].y = 0.0f;
		
	}
	
}

__host__
void startCuda(int numBoids) {
	printf("\nDefining cuda variables\n");
	dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));

	// Malloc for device
	hipMalloc((void**)&pos_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&vel_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&acc_dev, numBoids * sizeof(float2));

	hipMalloc((void**)&sep_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&align_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&cohesion_dev, numBoids * sizeof(float2));

	//malloc for host
	pos_host = (float2*)malloc(numBoids * sizeof(float2));
	vel_host = (float2*)malloc(numBoids * sizeof(float2));

	//set random velocity
	for (int i = 0; i < numBoids; i++) {
		vel_host[i].x = ((float) rand() / (RAND_MAX));
		vel_host[i].y = ((float) rand() / (RAND_MAX));
	}
	
	// Setup Kernels
	printf("\nGenerating initial position\n");
	generateInitialPosition<<<fullBlocksPerGrid, BlockSize>>>(numBoids, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);

	hipMemcpy(vel_dev, vel_host, numBoids * sizeof(float2), hipMemcpyHostToDevice);

	hipMemcpy(pos_host, pos_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);
	hipMemcpy(vel_host, vel_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);

	printf("after\n");
	for (int i = 0; i < numBoids; i++) {
		printf("x = %f, y = %f\n", vel_host[i].x, vel_host[i].y);
	}
}

__global__
void update(int numBoids, float2 averagePos, float2 averageForward, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		//cohesion
		float2 cohesion = calc_cohesion_accel(numBoids, averagePos, pos_dev);
		//separation
		//alignment
		
		vel_dev[i] = add2Vectors(vel_dev[i], cohesion);
		updatePos(numBoids, vel_dev, pos_dev);
	}
		
}

__host__
int main(int argc, char* argv[]) 
{
	dim3 fullBlocksPerGrid((int)ceil(float(NBOIDS) / float(BlockSize)));
	
	int iterations = 10;

  	startCuda(NBOIDS);
	for (int i = 0; i < iterations; i++) {
		calc_average_pos();
		calc_average_forward();
		update<<<fullBlocksPerGrid, BlockSize>>>(NBOIDS, averagePos, averageForward, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);

		//for debugging will remove
		hipMemcpy(vel_host, vel_dev, NBOIDS * sizeof(float2), hipMemcpyDeviceToHost);
		printf("\n---\nx: %f, y: %f", vel_host[0].x, vel_host[0].y);
	}

    
   hipFree(pos_dev);
   hipFree(vel_dev);
   hipFree(acc_dev);
   hipFree(sep_dev);
   hipFree(align_dev);
   hipFree(cohesion_dev);
	
   free(pos_host);
   return 0;
}
