
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <stdio.h>


//global variables
float2* pos_dev;
float2* vel_dev;
float2* acc_dev;

float2* sep_dev;
float2* align_dev;
float2* cohesion_dev;

float2* pos_host;
float2* vel_host;

float2  averagePos;
float2 averageForward;

#define BlockSize 256
#define NBOIDS 1000
#define FLOCKING_RAD 50.0f;
#define COHESION_STRENGTH 3.0f;
#define ALIGNMENT_STRENGTH 5.0f;
#define SEPARATION_STRENGTH 2.0f;
#define SAFE_RADIUS 3.0f;
#define MAX_SPEED 5.0f;

//vector math -- may update functions

__device__
bool vector2dEquals(float2 a, float2 b) {
	if (a.x == b.x && a.y == b.y) {
		return true;
	} else {
		return false;
	}
}

__device__
float distance(float2 myPos, float2 theirPos) {
	float dx = myPos.x - theirPos.x;
	float dy = myPos.y - theirPos.y;

	float dist = sqrt(dx*dx + dy*dy);
	return dist;
}


__device__
float2 add2dVectors(float2 v1, float2 v2) {
	float2 temp = make_float2(v1.x, v1.y);
	temp.x += v2.x;
	temp.y += v2.y;
	return temp;
}

__device__
float2 sub2dVectors(float2 v1, float2 v2) {
	float2 temp = make_float2(v1.x, v1.y);
	temp.x -= v2.x;
	temp.y -= v2.y;
	return temp;
}

__device__
float2 mulVectorByScalar(float scalar, float2 vector) {
	// Temp to not overwrite original vector
	float2 temp = make_float2(vector.x, vector.y);
	temp.x *= scalar;
	temp.y *= scalar;
	return temp;
}

__device__
float2 divVectorByScalar(float scalar, float2 vector) {
	float2 temp = make_float2(vector.x, vector.y);
	temp.x /= scalar;
	temp.y /= scalar;
	return temp;
}

__device__
float calcLength(float2 vec) {
	return sqrt(vec.x * vec.x + vec.y * vec.y
	);
}

__device__
float2 normalizeVector(float2 vector) {
	float2 temp = make_float2(vector.x, vector.y);
	float length = calcLength(temp);
	if (length > 0) {
		temp.x /= length;
		temp.y /= length;
	}
	return temp;
}

//-----------------end vec funcs------------------------

__host__ void calc_average_forward() {
	int counter = 0;
	float2 sum = make_float2(0.0, 0.0);
	for (int i = 0; i < NBOIDS; i++) {
		sum.x += vel_host[i].x;
		sum.y += vel_host[i].y;
		counter++;
	}
	averageForward.x = sum.x / counter;
	averageForward.y = sum.y / counter;
}

__host__ void calc_average_pos() {
	int counter = 0;
	float2 sum = make_float2(0.0, 0.0);
	for (int i = 0; i < NBOIDS; i++) {
		sum.x += pos_host[i].x;
		sum.y += pos_host[i].y;
		counter++;
	}
	averagePos.x = sum.x / counter;
	averagePos.y = sum.y / counter;
}

__global__ void updatePos(int numBoids, float2* vel_dev, float2* pos_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		pos_dev[i] = add2dVectors(pos_dev[i], vel_dev[i]);
		//pos_dev[i] = newPos;
	}
}

__device__ float2 calc_separation_accel(int numBoids, float2* pos_dev, float2* vel_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	float safeDist = SAFE_RADIUS;
	safeDist = safeDist + safeDist;
	float separationStrength = SEPARATION_STRENGTH;
	float2 totalVel = make_float2(0.0f, 0.0f);

	if (i < numBoids) { 
		float2 boidPos = make_float2(pos_dev[i].x, pos_dev[i].y);
		float2 boidVel = make_float2(vel_dev[i].x, vel_dev[i].y);

		for (int i = 0; i < numBoids; i++) {
			//printf("%s", vector2dEquals(boidPos, pos_dev[i]) && vector2dEquals(boidVel, vel_dev[i]) ? "true\n" : "");
			float2 siblingPos = pos_dev[i];
			float2 siblingVel = vel_dev[i];
			//check to see if current boid is self
			if (vector2dEquals(boidPos, siblingPos) && vector2dEquals(boidVel, siblingVel)) {
				continue;
			}

			float2 accel = sub2dVectors(boidPos, siblingPos);
			float dist = calcLength(accel);
			
			if (dist < safeDist) {
				accel = normalizeVector(accel);
				accel = divVectorByScalar(safeDist, mulVectorByScalar((safeDist - dist), accel));
				totalVel = add2dVectors(totalVel, accel);
			}
		}

		if (calcLength(totalVel) > 1) {
			totalVel = normalizeVector(totalVel);
		}

		return mulVectorByScalar(separationStrength, totalVel);
	}

	return make_float2(0.0f, 0.0f);
}

__device__ float2 calc_alignment_accel(int numBoids, float2 averageForward) {
	float maxSpeed = MAX_SPEED;
	float alignStr = ALIGNMENT_STRENGTH;

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		float2 accel = divVectorByScalar(maxSpeed, averageForward);
		if (calcLength(accel) > 1) {
			accel = normalizeVector(accel);
		}
		return mulVectorByScalar(alignStr, accel);
	}
	return make_float2(0.0f, 0.0f);
}

__device__ float2 calc_cohesion_accel(int numBoids, float2 averagePos, float2* pos_dev) {
	float flockRad = FLOCKING_RAD;
	float cohesionStr = COHESION_STRENGTH;

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		float2 accel = sub2dVectors(averagePos, pos_dev[i]);
		float dist = calcLength(pos_dev[i]);

		accel = normalizeVector(accel);

		if(dist < flockRad) {
			accel = mulVectorByScalar(dist, accel);
			accel = divVectorByScalar(flockRad, accel);
		}

		return mulVectorByScalar(cohesionStr, accel);
	}
	return make_float2(0.0f, 0.0f);
}

__global__
void generateInitialPosition(int numBoids, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		pos_dev[i].x = 0.0f;
		pos_dev[i].y = 0.0f;
		vel_dev[i].x = 0.0f;
		vel_dev[i].y = 0.0f;
		acc_dev[i].x = 0.0f;
		acc_dev[i].y = 0.0f;
		sep_dev[i].x = 0.0f;
		sep_dev[i].y = 0.0f;
		align_dev[i].x = 0.0f;
		align_dev[i].y = 0.0f;
		cohesion_dev[i].x = 0.0f;
		cohesion_dev[i].y = 0.0f;
		
	}
	
}

__host__
void startCuda(int numBoids) {
	printf("\nDefining cuda variables\n");
	dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));

	// Malloc for device
	hipMalloc((void**)&pos_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&vel_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&acc_dev, numBoids * sizeof(float2));

	hipMalloc((void**)&sep_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&align_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&cohesion_dev, numBoids * sizeof(float2));

	//malloc for host
	pos_host = (float2*)malloc(numBoids * sizeof(float2));
	vel_host = (float2*)malloc(numBoids * sizeof(float2));

	//set random velocity
	for (int i = 0; i < numBoids; i++) {
		vel_host[i].x = ((float) rand() / (RAND_MAX));
		vel_host[i].y = ((float) rand() / (RAND_MAX));
	}
	
	// Setup Kernels
	printf("\nGenerating initial position\n");
	generateInitialPosition<<<fullBlocksPerGrid, BlockSize>>>(numBoids, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);

	hipMemcpy(vel_dev, vel_host, numBoids * sizeof(float2), hipMemcpyHostToDevice);
	hipMemcpy(pos_host, pos_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);
	hipMemcpy(vel_host, vel_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);


	//for debugging
	/*printf("after\n");
	for (int i = 0; i < numBoids; i++) {
		printf("x = %f, y = %f\n", vel_host[i].x, vel_host[i].y);
	}*/
}

__global__
void update(int numBoids, float2 averagePos, float2 averageForward, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		//cohesion
		float2 cohesion = calc_cohesion_accel(numBoids, averagePos, pos_dev);
		//separation
		float2 separation = calc_separation_accel(numBoids, pos_dev, vel_dev);
		//alignment
		float2 alignment = calc_alignment_accel(numBoids, averageForward);
		
		//printf("cohesion: %f\nseparation: %f\nalignment: %f\n", cohesion, separation, alignment);

		vel_dev[i] = add2dVectors(vel_dev[i], cohesion);
		vel_dev[i] = add2dVectors(vel_dev[i], separation);
		vel_dev[i] = add2dVectors(vel_dev[i], alignment);
	}
		
}

__host__
int main(int argc, char* argv[]) 
{
	dim3 fullBlocksPerGrid((int)ceil(float(NBOIDS) / float(BlockSize)));
	
	int iterations = 1000;

  	startCuda(NBOIDS);

	printf("\nRunning Simulation with %d boids and %d iterations\n", NBOIDS, iterations);
	for (int i = 0; i < iterations; i++) {
		calc_average_pos();
		calc_average_forward();
		update<<<fullBlocksPerGrid, BlockSize>>>(NBOIDS, averagePos, averageForward, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);
		updatePos<<<fullBlocksPerGrid, BlockSize>>>(NBOIDS, vel_dev, pos_dev);
		//for debugging will remove
		//cudaMemcpy(vel_host, vel_dev, NBOIDS * sizeof(float2), cudaMemcpyDeviceToHost);
		//cudaMemcpy(pos_host, pos_dev, NBOIDS * sizeof(float2), cudaMemcpyDeviceToHost);
		//printf("guy1-x: %f, guy1-y: %f | ", pos_host[0].x, pos_host[0].y);
		//printf("guy2-x: %f, guy2-y: %f\n", pos_host[1].x, pos_host[1].y);
	}

    
   hipFree(pos_dev);
   hipFree(vel_dev);
   hipFree(acc_dev);
   hipFree(sep_dev);
   hipFree(align_dev);
   hipFree(cohesion_dev);
	
   free(pos_host);
   return 0;
}