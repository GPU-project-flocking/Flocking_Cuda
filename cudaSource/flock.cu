//These includes are for running on a personal computer
/*#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <cuda_runtime_api.h>
#include <stdio.h>
#include <device_functions.h>
#include <cuda.h>
#include <crt/host_defines.h>
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <string>
#include <chrono>


//global variables definitions for the boids on both device and host
float2* pos_dev;
float2* vel_dev;
float2* acc_dev;

float2* sep_dev;
float2* align_dev;
float2* cohesion_dev;

float2* pos_host;
float2* vel_host;

float2 averagePos;
float2 averageForward;

//all of our hard coded values we can change
#define BlockSize 256
#define FLOCKING_RAD 50.0f
#define COHESION_STRENGTH 3.0f
#define ALIGNMENT_STRENGTH 5.0f
#define SEPARATION_STRENGTH 2.0f
#define SAFE_RADIUS 3.0f
#define MAX_SPEED 5.0f

//vector math functions for the 2d vectors -- naive

__device__ bool vector2dEquals(float2 a, float2 b) {
	if (a.x == b.x && a.y == b.y) {
		return true;
	} else {
		return false;
	}
}

__device__ float calcLength(float2 vec) {
	return sqrt(vec.x * vec.x + vec.y * vec.y);
}

__device__ float distance(float2 vec1, float2 vec2) {
	float finalDistance = sqrt(((vec1.x - vec2.x)*(vec1.x - vec2.x)) + ((vec1.y - vec2.y)*(vec1.y - vec2.y)));	
	return finalDistance;
}

__device__ float2 subVecs(float2 vec1, float2 vec2) {
	float2 finalVec = make_float2(vec1.x - vec2.x, vec1.y - vec2.y);
	return finalVec;
}

__device__ float2 addVecs(float2 vec1, float2 vec2) {
	float2 finalVec = make_float2(vec1.x + vec2.x, vec1.y + vec2.y);
	return finalVec;
}

__device__ float2 divideVec(float scalar, float2 vector) {
	float2 finalVec = make_float2(vector.x / scalar, vector.y / scalar);
	return finalVec;
}

__device__ float2 multiplyVec(float scalar, float2 vector) {
	float2 finalVec = make_float2(vector.x * scalar, vector.y * scalar);
	return finalVec;
}

__device__ float2 normalize(float2 vector) {
	float length = calcLength(vector);
	if (length > 0) {
		float2 finalVec = make_float2(vector.x / length, vector.y / length);
		return finalVec;
	} else {
		return vector;
	}
}

//-----------------end vec funcs------------------------


// calculates the average forward velocity vector of all the boids
__host__ void calc_average_forward(int NBOIDS) {
	int counter = 0;
	float2 sum = make_float2(0.0, 0.0);
	for (int i = 0; i < NBOIDS; i++) {
		sum.x += vel_host[i].x;
		sum.y += vel_host[i].y;
		counter++;
	}
	averageForward.x = sum.x / counter;
	averageForward.y = sum.y / counter;
}

// calculate the average position of all the boids
__host__ void calc_average_pos(int NBOIDS) {
	int counter = 0;
	float2 sum = make_float2(0.0, 0.0);
	for (int i = 0; i < NBOIDS; i++) {
		sum.x += pos_host[i].x;
		sum.y += pos_host[i].y;
		counter++;
	}
	averagePos.x = sum.x / counter;
	averagePos.y = sum.y / counter;
}

//updates the position of all the boids
__global__ void updatePos(int numBoids, float2* vel_dev, float2* pos_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	//if boids get too far away set their position to 0;
	if (i < numBoids) {
		if (pos_dev[i].x > 10000.0f || pos_dev[i].y > 10000.0f) {
			pos_dev[i].x = 0;
			pos_dev[i].y = 0;
		}

		pos_dev[i] = addVecs(pos_dev[i], vel_dev[i]);
	}
}

//calculates the separation vector for each boid
__device__ float2 calc_separation_accel(int numBoids, float2* pos_dev, float2* vel_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	float safeDist = SAFE_RADIUS;
	safeDist = safeDist + safeDist;
	float separationStrength = SEPARATION_STRENGTH;
	float2 totalVel = make_float2(0.0f, 0.0f);

	if (i < numBoids) { 
		float2 boidPos = make_float2(pos_dev[i].x, pos_dev[i].y);
		float2 boidVel = make_float2(vel_dev[i].x, vel_dev[i].y);

		for (int i = 0; i < numBoids; i++) {
			float2 siblingPos = pos_dev[i];
			float2 siblingVel = vel_dev[i];
			//skip if current boid is self
			if (vector2dEquals(boidPos, siblingPos) && vector2dEquals(boidVel, siblingVel)) {
				continue;
			}

			float2 accel = subVecs(boidPos, siblingPos);
			float dist = calcLength(accel);
			if (dist < safeDist) {
				accel = normalize(accel);
				accel = divideVec(safeDist, multiplyVec((safeDist - dist), accel));
				totalVel = addVecs(totalVel, accel);
			}
		}

		if (calcLength(totalVel) > 1) {
			totalVel = normalize(totalVel);
		}

		return multiplyVec(separationStrength, totalVel);
	}

	return make_float2(0.0f, 0.0f);
}

//calculates the alignment vector for each boid
__device__ float2 calc_alignment_accel(int numBoids, float2 averageForward) {
	float maxSpeed = MAX_SPEED;
	float alignStr = ALIGNMENT_STRENGTH;

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		float2 accel = divideVec(maxSpeed, averageForward);
		if (calcLength(accel) > 1) {
			accel = normalize(accel);
		}
		return multiplyVec(alignStr, accel);
	}
	return make_float2(0.0f, 0.0f);
}

//calculates the cohesion vector for each boid
__device__ float2 calc_cohesion_accel(int numBoids, float2 averagePos, float2* pos_dev) {
	float flockRad = FLOCKING_RAD;
	float cohesionStr = COHESION_STRENGTH;

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		float2 accel = subVecs(averagePos, pos_dev[i]);
		float dist = calcLength(pos_dev[i]);

		accel = normalize(accel);

		if(dist < flockRad) {
			accel = multiplyVec(dist, accel);
			accel = divideVec(flockRad, accel);
		}

		return multiplyVec(cohesionStr, accel);
	}
	return make_float2(0.0f, 0.0f);
}

//generates the initial position of the boids
__global__ void generateInitialPosition(int numBoids, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		pos_dev[i].x = 0.0f;
		pos_dev[i].y = 0.0f;
		vel_dev[i].x = 0.0f;
		vel_dev[i].y = 0.0f;
		acc_dev[i].x = 0.0f;
		acc_dev[i].y = 0.0f;
		sep_dev[i].x = 0.0f;
		sep_dev[i].y = 0.0f;
		align_dev[i].x = 0.0f;
		align_dev[i].y = 0.0f;
		cohesion_dev[i].x = 0.0f;
		cohesion_dev[i].y = 0.0f;
		
	}
	
}

//define inital cuda mallocs and vars
__host__ void startCuda(int numBoids) {
	//printf("\nDefining cuda variables\n");
	dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));

	// Malloc for device
	hipMalloc((void**)&pos_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&vel_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&acc_dev, numBoids * sizeof(float2));

	hipMalloc((void**)&sep_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&align_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&cohesion_dev, numBoids * sizeof(float2));

	//malloc for host
	pos_host = (float2*)malloc(numBoids * sizeof(float2));
	vel_host = (float2*)malloc(numBoids * sizeof(float2));

	//set random velocity
	for (int i = 0; i < numBoids; i++) {
		vel_host[i].x = ((float) rand() / (RAND_MAX));
		vel_host[i].y = ((float) rand() / (RAND_MAX));
	}
	
	// Setup Kernels
	//printf("\nGenerating initial position\n");
	generateInitialPosition<<<fullBlocksPerGrid, BlockSize>>>(numBoids, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);

	hipMemcpy(vel_dev, vel_host, numBoids * sizeof(float2), hipMemcpyHostToDevice);
	hipMemcpy(pos_host, pos_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);
	hipMemcpy(vel_host, vel_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);


	//for debugging
	/*printf("after\n");
	for (int i = 0; i < numBoids; i++) {
		printf("x = %f, y = %f\n", vel_host[i].x, vel_host[i].y);
	}*/
}

//update kernel that calls cohesion, separation and alignment
__global__ void update(int numBoids, float2 averagePos, float2 averageForward, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		//cohesion
		float2 cohesion = calc_cohesion_accel(numBoids, averagePos, pos_dev);
		//separation
		float2 separation = calc_separation_accel(numBoids, pos_dev, vel_dev);
		//alignment
		float2 alignment = calc_alignment_accel(numBoids, averageForward);
		
		//printf("cohesion: %f\nseparation: %f\nalignment: %f\n", cohesion, separation, alignment);

		vel_dev[i] = addVecs(vel_dev[i], cohesion);
		vel_dev[i] = addVecs(vel_dev[i], separation);
		vel_dev[i] = addVecs(vel_dev[i], alignment);

		if (calcLength(vel_dev[i]) > 50.0f) {
			vel_dev[i] = normalize(vel_dev[i]);
			vel_dev[i] = multiplyVec(50.0f, vel_dev[i]);
			//printf("%d ", calcLength(vel_dev[i]));	
		}
		if (calcLength(vel_dev[i]) < 0.0f) {
			vel_dev[i] = normalize(vel_dev[i]);
			vel_dev[i] = multiplyVec(50.0f, vel_dev[i]);
			//printf("%d ", calcLength(vel_dev[i]));	
		}
		//printf("%d ", calcLength(vel_dev[i]));	
	}
		
}

//main cuda function
__host__ int main(int argc, char* argv[]) 
{	
	//for timing
	using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

    auto t1 = high_resolution_clock::now();

	//takes 2 arguments, number of boids and iterations
	int numB = std::stoi(argv[1]);
	int iterations = std::stoi(argv[2]);

	

	dim3 fullBlocksPerGrid((int)ceil(float(numB) / float(BlockSize)));
	

  	startCuda(numB);

	//printf("\nRunning Simulation with %d boids and %d iterations\n", numB, iterations);
	for (int i = 0; i < iterations; i++) {
		
		hipMemcpy(vel_host, vel_dev, numB * sizeof(float2), hipMemcpyDeviceToHost);
		hipMemcpy(pos_host, pos_dev, numB * sizeof(float2), hipMemcpyDeviceToHost);
		calc_average_pos(numB);
		calc_average_forward(numB);
		
		
		update<<<fullBlocksPerGrid, BlockSize>>>(numB, averagePos, averageForward, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);
		updatePos<<<fullBlocksPerGrid, BlockSize>>>(numB, vel_dev, pos_dev);
		//for debugging will remove
		//cudaMemcpy(vel_host, vel_dev, numB * sizeof(float2), cudaMemcpyDeviceToHost);
		//cudaMemcpy(pos_host, pos_dev, numB * sizeof(float2), cudaMemcpyDeviceToHost);
		//printf("guy1-x: %f, guy1-y: %f | ", pos_host[0].x, pos_host[0].y);
		//printf("guy2-x: %f, guy2-y: %f\n", pos_host[1].x, pos_host[1].y);
	}

    
   hipFree(pos_dev);
   hipFree(vel_dev);
   hipFree(acc_dev);
   hipFree(sep_dev);
   hipFree(align_dev);
   hipFree(cohesion_dev);
	
   free(pos_host);

   auto t2 = high_resolution_clock::now();
   duration<double, std::milli> ms_double = t2 - t1;
   printf("%f", ms_double);

   return 0;
}
