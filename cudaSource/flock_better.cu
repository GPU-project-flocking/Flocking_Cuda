#include "hip/hip_runtime.h"
//These includes are for running on a personal computer
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>

#include "flock_better.cuh"
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <string>
#include <chrono>




//global variables definitions for the boids on both device and host
float* pos_tmp_dev_x;
float* pos_tmp_dev_y;

float* vel_tmp_dev_x;
float* vel_tmp_dev_y;

float2* pos_dev;
float2* vel_dev;
float2* acc_dev;

float2* sep_dev;
float2* align_dev;
float2* cohesion_dev;

float2* pos_host;
float2* vel_host;

float2 averagePos;
float2 averageForward;

//all of our hard coded values we can change
#define BlockSize 256
#define FLOCKING_RAD 50.0f
#define COHESION_STRENGTH 3.0f
#define ALIGNMENT_STRENGTH 5.0f
#define SEPARATION_STRENGTH 2.0f
#define SAFE_RADIUS 3.0f
#define MAX_SPEED 5.0f

//vector math functions for the 2d vectors -- naive

__device__ bool vector2dEquals(float2 a, float2 b) {
	if (a.x == b.x && a.y == b.y) {
		return true;
	} else {
		return false;
	}
}

__device__ float calcLength(float2 vec) {
	return sqrt(vec.x * vec.x + vec.y * vec.y);
}

__device__ float distance(float2 vec1, float2 vec2) {
	float finalDistance = sqrt(((vec1.x - vec2.x)*(vec1.x - vec2.x)) + ((vec1.y - vec2.y)*(vec1.y - vec2.y)));	
	return finalDistance;
}

__device__ float2 subVecs(float2 vec1, float2 vec2) {
	float2 finalVec = make_float2(vec1.x - vec2.x, vec1.y - vec2.y);
	return finalVec;
}

__device__ float2 addVecs(float2 vec1, float2 vec2) {
	float2 finalVec = make_float2(vec1.x + vec2.x, vec1.y + vec2.y);
	return finalVec;
}

__device__ float2 divideVec(float scalar, float2 vector) {
	float2 finalVec = make_float2(vector.x / scalar, vector.y / scalar);
	return finalVec;
}

__device__ float2 multiplyVec(float scalar, float2 vector) {
	float2 finalVec = make_float2(vector.x * scalar, vector.y * scalar);
	return finalVec;
}

__device__ float2 normalize(float2 vector) {
	float length = calcLength(vector);
	if (length > 0) {
		float2 finalVec = make_float2(vector.x / length, vector.y / length);
		return finalVec;
	} else {
		return vector;
	}
}

//-----------------end vec funcs------------------------


// used this github as inspiration for this summation reduction kernel:  https://github.com/mark-poscablo/gpu-sum-reduction/blob/master/sum_reduction/reduce.cu#L196
__global__ void calc_average_forw_and_pos_device(int numBoids, float2* vel_arr_dev, float2 vel_dev, float2* pos_arr_dev,  float2 pos_dev,float* pos_tmp_dev_x, float* pos_tmp_dev_y, float* vel_tmp_dev_x, float* vel_tmp_dev_y) {

	extern __shared__ float s_sum_pos_x[];
	extern __shared__ float s_sum_pos_y[];
	extern __shared__ float s_sum_vel_x[];
	extern __shared__ float s_sum_vel_y[];
	
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	unsigned int thr_idx = threadIdx.x;
	
	s_sum_vel_x[thr_idx] = 0;
	s_sum_vel_y[thr_idx] = 0;
	s_sum_pos_x[thr_idx] = 0;
	s_sum_pos_y[thr_idx] = 0;

	/*if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		pos_tmp_dev_x = 0;
		pos_tmp_dev_y = 0;
		vel_tmp_dev_x = 0;
		vel_tmp_dev_y = 0;
		
	}*/
	if(i < numBoids)
	{
		s_sum_vel_x[thr_idx] = vel_arr_dev[i].x + vel_arr_dev[i + blockDim.x].x;
		s_sum_vel_y[thr_idx] = vel_arr_dev[i].y + vel_arr_dev[i + blockDim.x].y;
	
		
		s_sum_pos_x[thr_idx] = pos_arr_dev[i].x + pos_arr_dev[i + blockDim.x].x;
		s_sum_pos_y[thr_idx] = pos_arr_dev[i].y + pos_arr_dev[i + blockDim.x].y;
	}
	
	__syncthreads();
	

	for (int x = blockDim.x; x > 0; x >>= 1) {
	
		
	
		if(thr_idx < x)
		{
			s_sum_vel_x[thr_idx] += s_sum_vel_x[thr_idx + x];
			s_sum_vel_y[thr_idx] += s_sum_vel_y[thr_idx + x];
	
			s_sum_pos_x[thr_idx] += s_sum_pos_x[thr_idx + x];
			s_sum_pos_y[thr_idx] += s_sum_pos_y[thr_idx + x];
		}
	
		__syncthreads();
	}
	

	 if (thr_idx == 0)
	 {
		
		 atomicAdd(vel_tmp_dev_x, s_sum_vel_x[0]);
		 atomicAdd(vel_tmp_dev_y, s_sum_vel_y[0]);
		 atomicAdd(pos_tmp_dev_x, s_sum_pos_x[0]);
		 atomicAdd(pos_tmp_dev_y, s_sum_pos_y[0]);
	

		 
	 }

	//hipDeviceSynchronize();

	 if (threadIdx.x == 0 && blockIdx.x == 0)
	 {
	 	vel_dev.x = *vel_tmp_dev_x / numBoids;
	 	vel_dev.y = *vel_tmp_dev_y / numBoids;
	 	pos_dev.x = *pos_tmp_dev_x / numBoids;
	 	pos_dev.y = *pos_tmp_dev_y / numBoids;
		printf("test");
	 	//printf("avg pos device is: %d \n", vel_dev.x);
	 }
	 //printf("test");
	
}


//updates the position of all the boids
__global__ void updatePos(int numBoids, float2* vel_dev, float2* pos_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	//if boids get too far away set their position to 0;
	if (i < numBoids) {
		if (pos_dev[i].x > 10000.0f || pos_dev[i].y > 10000.0f) {
			pos_dev[i].x = 0;
			pos_dev[i].y = 0;
		}

		pos_dev[i] = addVecs(pos_dev[i], vel_dev[i]);
	}
}

//calculates the separation vector for each boid
__device__ float2 calc_separation_accel(int numBoids, float2* pos_dev, float2* vel_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	float safeDist = SAFE_RADIUS;
	safeDist = safeDist + safeDist;
	float separationStrength = SEPARATION_STRENGTH;
	float2 totalVel = make_float2(0.0f, 0.0f);

	if (i < numBoids) { 
		float2 boidPos = make_float2(pos_dev[i].x, pos_dev[i].y);
		float2 boidVel = make_float2(vel_dev[i].x, vel_dev[i].y);

		for (int i = 0; i < numBoids; i++) {
			float2 siblingPos = pos_dev[i];
			float2 siblingVel = vel_dev[i];
			//skip if current boid is self
			if (vector2dEquals(boidPos, siblingPos) && vector2dEquals(boidVel, siblingVel)) {
				continue;
			}

			float2 accel = subVecs(boidPos, siblingPos);
			float dist = calcLength(accel);

			if (dist > 8.0f) {
				continue;
			}

			if (dist < safeDist) {
				accel = normalize(accel);
				accel = divideVec(safeDist, multiplyVec((safeDist - dist), accel));
				totalVel = addVecs(totalVel, accel);
			}
		}

		if (calcLength(totalVel) > 1) {
			totalVel = normalize(totalVel);
		}

		return multiplyVec(separationStrength, totalVel);
	}

	return make_float2(0.0f, 0.0f);
}


//calculates the alignment vector for each boid
__device__ float2 calc_alignment_accel(int numBoids, float2 averageForward) {
	float maxSpeed = MAX_SPEED;
	float alignStr = ALIGNMENT_STRENGTH;

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		float2 accel = divideVec(maxSpeed, averageForward);
		if (calcLength(accel) > 1) {
			accel = normalize(accel);
		}
		return multiplyVec(alignStr, accel);
	}
	return make_float2(0.0f, 0.0f);
}

//calculates the cohesion vector for each boid
__device__ float2 calc_cohesion_accel(int numBoids, float2 averagePos, float2* pos_dev) {
	float flockRad = FLOCKING_RAD;
	float cohesionStr = COHESION_STRENGTH;

	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		float2 accel = subVecs(averagePos, pos_dev[i]);
		float dist = calcLength(pos_dev[i]);

		accel = normalize(accel);

		if(dist < flockRad) {
			accel = multiplyVec(dist, accel);
			accel = divideVec(flockRad, accel);
		}

		return multiplyVec(cohesionStr, accel);
	}
	return make_float2(0.0f, 0.0f);
}

__global__
void generateInitialPosition(int numBoids, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numBoids) {
		pos_dev[i].x = 0.0f;
		pos_dev[i].y = 0.0f;
		vel_dev[i].x = 0.0f;
		vel_dev[i].y = 0.0f;
		acc_dev[i].x = 0.0f;
		acc_dev[i].y = 0.0f;
		sep_dev[i].x = 0.0f;
		sep_dev[i].y = 0.0f;
		align_dev[i].x = 0.0f;
		align_dev[i].y = 0.0f;
		cohesion_dev[i].x = 0.0f;
		cohesion_dev[i].y = 0.0f;
		
		
	}
	
}

__host__ void startCuda(int numBoids) {
	//printf("\nDefining cuda variables\n");
	dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));

	// Malloc for device
	hipMalloc((void**)&pos_tmp_dev_x,sizeof(float));
	hipMalloc((void**)&pos_tmp_dev_y,sizeof(float));

	hipMalloc((void**)&vel_tmp_dev_x, sizeof(float));
	hipMalloc((void**)&vel_tmp_dev_y,sizeof(float));
	
	hipMalloc((void**)&pos_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&vel_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&acc_dev, numBoids * sizeof(float2));

	hipMalloc((void**)&sep_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&align_dev, numBoids * sizeof(float2));
	hipMalloc((void**)&cohesion_dev, numBoids * sizeof(float2));

	hipMalloc((void**)&averagePos, sizeof(float2));
	hipMalloc((void**)&averageForward, sizeof(float2));

	//malloc for host
	pos_host = (float2*)malloc(numBoids * sizeof(float2));
	vel_host = (float2*)malloc(numBoids * sizeof(float2));

	//set random velocity
	for (int i = 0; i < numBoids; i++) {
		vel_host[i].x = ((float) rand() / (RAND_MAX));
		vel_host[i].y = ((float) rand() / (RAND_MAX));
	}
	
	// Setup Kernels
	//printf("\nGenerating initial position\n");

	

	generateInitialPosition<<<fullBlocksPerGrid, BlockSize>>>(numBoids, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);

	hipMemcpy(vel_dev, vel_host, numBoids * sizeof(float2), hipMemcpyHostToDevice);
	hipMemcpy(pos_host, pos_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);
	hipMemcpy(vel_host, vel_dev, numBoids * sizeof(float2), hipMemcpyDeviceToHost);


	//for debugging
	/*printf("after\n");
	for (int i = 0; i < numBoids; i++) {
		printf("x = %f, y = %f\n", vel_host[i].x, vel_host[i].y);
	}*/
}

//update kernel that calls cohesion, separation and alignment
__global__ void update(int numBoids, float2 averagePos, float2 averageForward, float2* pos_dev, float2* vel_dev, float2* acc_dev, float2* sep_dev, float2* align_dev, float2* cohesion_dev) {
	//dim3 fullBlocksPerGrid((int)ceil(float(numBoids) / float(BlockSize)));
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < numBoids) {
		//cohesion
		float2 cohesion = calc_cohesion_accel(numBoids, averagePos, pos_dev);
		//separation
		float2 separation = calc_separation_accel(numBoids, pos_dev, vel_dev);
		//alignment
		float2 alignment = calc_alignment_accel(numBoids, averageForward);
		
		//printf("cohesion: %f\nseparation: %f\nalignment: %f\n", cohesion, separation, alignment);

		vel_dev[i] = addVecs(vel_dev[i], cohesion);
		vel_dev[i] = addVecs(vel_dev[i], separation);
		vel_dev[i] = addVecs(vel_dev[i], alignment);

		if (calcLength(vel_dev[i]) > 50.0f) {
			vel_dev[i] = normalize(vel_dev[i]);
			vel_dev[i] = multiplyVec(50.0f, vel_dev[i]);
			//printf("%d ", calcLength(vel_dev[i]));	
		}
		if (calcLength(vel_dev[i]) < 0.0f) {
			vel_dev[i] = normalize(vel_dev[i]);
			vel_dev[i] = multiplyVec(50.0f, vel_dev[i]);
			//printf("%d ", calcLength(vel_dev[i]));	
		}
		//printf("%d ", calcLength(vel_dev[i]));	
	}
		
}
//
//
//__host__
//int main(int argc, char* argv[]) 
//{
//	using std::chrono::high_resolution_clock;
//    using std::chrono::duration_cast;
//    using std::chrono::duration;
//    using std::chrono::milliseconds;
//
//    auto t1 = high_resolution_clock::now();
//
//	/*int numB = std::stoi(argv[1]);
//	int iterations = std::stoi(argv[2]);*/
//
//	int numB = 1000;
//	int iterations = 1000;
//	
//	dim3 fullBlocksPerGrid((int)ceil(float(numB) / float(BlockSize)));
//
//  	startCuda(numB);
//
//	//printf("\nRunning Simulation with %d boids and %d iterations\n", numB, iterations);
//	for (int i = 0; i < iterations; i++) {
//
//		hipMemcpy(pos_tmp_dev_x, 0, sizeof(float), hipMemcpyHostToDevice);
//		hipMemcpy(pos_tmp_dev_y, 0, sizeof(float), hipMemcpyHostToDevice);
//		hipMemcpy(vel_tmp_dev_x, 0, sizeof(float), hipMemcpyHostToDevice);
//		hipMemcpy(vel_tmp_dev_y, 0, sizeof(float), hipMemcpyHostToDevice);
//		calc_average_forw_and_pos_device<< <fullBlocksPerGrid, BlockSize, numB >> > (numB, vel_dev, averageForward, pos_dev, averagePos, pos_tmp_dev_x, pos_tmp_dev_y, vel_tmp_dev_x, vel_tmp_dev_y);
//
//		update<<<fullBlocksPerGrid, BlockSize>>>(numB, averagePos, averageForward, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);
//		updatePos<<<fullBlocksPerGrid, BlockSize>>>(numB, vel_dev, pos_dev);
//		//for debugging will remove
//		//hipMemcpy(vel_host, vel_dev, numB * sizeof(float2), hipMemcpyDeviceToHost);
//		//hipMemcpy(pos_host, pos_dev, numB * sizeof(float2), hipMemcpyDeviceToHost);
//		//printf("guy1-x: %f, guy1-y: %f | ", pos_host[0].x, pos_host[0].y);
//		//printf("guy2-x: %f, guy2-y: %f\n", pos_host[1].x, pos_host[1].y);
//	}
//
//    
//   hipFree(pos_dev);
//   hipFree(vel_dev);
//   hipFree(acc_dev);
//   hipFree(sep_dev);
//   hipFree(align_dev);
//   hipFree(cohesion_dev);
//	
//   free(pos_host);
//
//   auto t2 = high_resolution_clock::now();
//   duration<double, std::milli> ms_double = t2 - t1;
//   printf("%f", ms_double);
//
//   return 0;
//}
__host__
void testing_cuda()
{
	printf("testinf from cuda");
}

__host__
int setup_flock_cuda(int numB)
{
	

	/*int numB = std::stoi(argv[1]);
	int iterations = std::stoi(argv[2]);*/


	dim3 fullBlocksPerGrid((int)ceil(float(numB) / float(BlockSize)));

	startCuda(numB);

	//printf("\nRunning Simulation with %d boids and %d iterations\n", numB, iterations);
	


	

	return 0;
}

__host__
void update_flock_cuda(int numB, float2* vel_host, float2* pos_host)
{
	dim3 fullBlocksPerGrid((int)ceil(float(numB) / float(BlockSize)));

	hipMemcpy(pos_tmp_dev_x, 0, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pos_tmp_dev_y, 0, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(vel_tmp_dev_x, 0, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(vel_tmp_dev_y, 0, sizeof(float), hipMemcpyHostToDevice);
	calc_average_forw_and_pos_device << <fullBlocksPerGrid, BlockSize, numB >> > (numB, vel_dev, averageForward, pos_dev, averagePos, pos_tmp_dev_x, pos_tmp_dev_y, vel_tmp_dev_x, vel_tmp_dev_y);

	update << <fullBlocksPerGrid, BlockSize >> > (numB, averagePos, averageForward, pos_dev, vel_dev, acc_dev, sep_dev, align_dev, cohesion_dev);
	updatePos << <fullBlocksPerGrid, BlockSize >> > (numB, vel_dev, pos_dev);

	hipMemcpy(vel_host, vel_dev, numB * sizeof(float2), hipMemcpyDeviceToHost);
	hipMemcpy(pos_host, pos_dev, numB * sizeof(float2), hipMemcpyDeviceToHost);

	
}

__host__
void free_flock_cuda()
{
	hipFree(pos_dev);
	hipFree(vel_dev);
	hipFree(acc_dev);
	hipFree(sep_dev);
	hipFree(align_dev);
	hipFree(cohesion_dev);

	free(pos_host);

	
}

